#include "hip/hip_runtime.h"
#include "dbscan.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stack>
#include <time.h>
#include <ctime>

__constant__ Point sp[2000];
 
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }


//####################################################################################

float __device__ cudaManhattanDistance(const Point &source, const Point &destination)	//the Manhattan distance for 128D points
{
	float dist = 0.0;

	for (unsigned i=0; i < 128; i++)
		dist += fabs(source.a[i] - destination.a[i]);
	
	return dist;
}

float __device__ cudaEuclideanDistance(const Point &source, const Point &destination)	//the Euclidean distance for 128D points
{
 	float dist = 0.0;

 	for (unsigned i=0; i < 128; i++)
		dist += pow(source.a[i] - destination.a[i], 2);

	return sqrt(dist);
}


float __device__ cudaManhattanDistance2D(const Point2D &source, const Point2D &destination)
{
	return (fabs(source.x - destination.x) + fabs(source.y - destination.y)	);
}

float __device__ cudaEuclideanDistance2D(const Point2D &source, const Point2D &destination)
{
 	return sqrt(	pow(source.x - destination.x, 2) + pow(source.y - destination.y, 2)	);
}

void __global__ cudaCreateNeighborsTableShared(Point * inputPoints, int pointNumber,
		float eps, int * cudaNeighborhoodMatrix, int minPoint)					//the version with shared memory
{
	unsigned int id;
	unsigned int source;
	unsigned int destination;
	unsigned int symValue;
	float dist;
	int sum;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	id = tid;
	
	while(id < pointNumber * pointNumber) // neighborhood matrix
	{
		source = id / pointNumber;		// device x
		destination = id % pointNumber; // device y

		if(source < destination)
		{
			symValue = destination * pointNumber + source;
			
			dist = 0.0;

 			for (unsigned i = 0; i < 128; i++)
				dist += pow(sp[source].a[i] - sp[destination].a[i], 2);  //euclidean distance (faster than calling the function)

			//dist = sqrt(	pow(sp[source].x - sp[destination].x, 2) + pow(sp[source].y - sp[destination].y, 2)	);
			if(dist < eps) //&& cudaDistTable[id] != 0)
			{
				cudaNeighborhoodMatrix[id] = 1;
				cudaNeighborhoodMatrix[symValue] = 1;
			}
			else
			{
				cudaNeighborhoodMatrix[id] = 0;
				cudaNeighborhoodMatrix[symValue] = 0;
			}
		}

		id += blockDim.x * gridDim.x;
	}

	__syncthreads();
	
	id = tid;
	while(id < pointNumber) // tag core
	{
		sum = 0;
		source = id * pointNumber;
		for (int i = 0; i < pointNumber;i++)
		{
			if(id != i) // not itself
			{
				if(cudaNeighborhoodMatrix[source+i])
				{
					cudaNeighborhoodMatrix[sum] = i;
					sum++;
				}
			}
		}
		
		if(sum >= minPoint)
		{
			inputPoints[id].hint = 1; //hint = 1 - core;
		}
		id += blockDim.x * gridDim.x;
	}
}


void __global__ cudaCreateNeighborsTable(Point * inputPoints, int pointNumber,
		float eps, int * cudaNeighborhoodMatrix, int minPoint)
{
	unsigned int id;
	unsigned int source;
	unsigned int destination;
	unsigned int symValue;
	int sum;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	id = tid;
	
	while(id < pointNumber * pointNumber) // neighborhood matrix
	{
		source = id / pointNumber;		// device x
		destination = id % pointNumber; // device y
		
		if(source < destination)
		{
			symValue = destination * pointNumber + source;

			if(cudaEuclideanDistance(inputPoints[source], inputPoints[destination]) < eps) //&& cudaDistTable[id] != 0)
			{
				cudaNeighborhoodMatrix[id] = 1;
				cudaNeighborhoodMatrix[symValue] = 1;
			}
			else
			{
				cudaNeighborhoodMatrix[id] = 0;
				cudaNeighborhoodMatrix[symValue] = 0;
			}
		}
		id += blockDim.x * gridDim.x;
	}
	
	__syncthreads();
	
	id = tid;
	while(id < pointNumber) // tag core
	{
		sum = 0;
		source = id * pointNumber;
		for(int i = 0; i < pointNumber; i++)
		{
			if(id != i) // not itself
			{
				if(cudaNeighborhoodMatrix[source+i])
				{
					cudaNeighborhoodMatrix[sum] = i;
					sum++;
				}
			}
		}
		if(sum>=minPoint)
		{
			inputPoints[id].hint = 1; //hint = 1 - core;
		}
		id += blockDim.x * gridDim.x;
	}
}

void __global__ cudaLabeling(Point * inputPoints, int pointNumber, float * cudaPowerMatrix)
{
	unsigned int id;
	unsigned int source;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	id = tid;
	
	while(id < pointNumber)
	{
		source = id * pointNumber;
		for(int i = 0; i < pointNumber; i++)
		{
			if(cudaPowerMatrix[source + i] > 0)
			{
				inputPoints[id].label = i;
				cudaPowerMatrix[source + i] = 0;
				break;
			}
		}
		id += blockDim.x * gridDim.x;
	}
}

void hostLabeling(Point *inputPoints, int pointNumber, int *hostNeighborhoodMatrix)
{
	std::stack<int> stack;
	int label = 1;

	for (int i = 0; i < pointNumber; i++)
	{
		if(inputPoints[i].hint == 1) // if the point is a core point
		{
			inputPoints[i].label = label;
			int offset = pointNumber*i;
			
			while(hostNeighborhoodMatrix[offset] != 0)
			{
 				inputPoints[hostNeighborhoodMatrix[offset]].label = label;
 				stack.push(hostNeighborhoodMatrix[offset]);
				offset++;
			}

			while(!stack.empty())
			{
				if(inputPoints[stack.top()].hint == 1)
				{
					offset = stack.top() * pointNumber;
					while(hostNeighborhoodMatrix[offset] != 0)
					{
						if(inputPoints[hostNeighborhoodMatrix[offset]].label < 1)
						{
							inputPoints[hostNeighborhoodMatrix[offset]].label = label;
							stack.push(hostNeighborhoodMatrix[offset]);
						}
						offset++;
					}
				}
				stack.pop();
			}

			label++;
		}
	}
}

Point *cudaDbscanNR(Point *inputPoints, int pointNumber, float eps, int minPoint)
{
	hipEvent_t start, stop, allocate, memcopy, neighbor,  labeling, mainCopy, sstop;
	hipEventCreate(&start);
	hipEventCreate(&allocate);
	hipEventCreate(&memcopy);
	hipEventCreate(&neighbor);
	hipEventCreate(&labeling);
	hipEventCreate(&mainCopy);
	hipEventCreate(&stop);
	hipEventCreate(&sstop);
	hipEventRecord( start, 0 );

	int hostLabelingTime;
	int *hostNeighborhoodMatrix = (int*) malloc(pointNumber*pointNumber*sizeof(int));

	Point *cudaPoints;
	int *cudaNeighborhoodMatrix;

	int allNum = pointNumber * pointNumber;
	int threadsNum = 128;

//############################################################
	CUDA_CHECK_RETURN(hipMalloc((void**)&cudaPoints, pointNumber * sizeof(Point)));

	CUDA_CHECK_RETURN(hipMalloc((int**)&cudaNeighborhoodMatrix, pointNumber * pointNumber * sizeof(int)));

	hipEventRecord( allocate, 0 );

//############################################################
	CUDA_CHECK_RETURN(hipMemcpy(cudaPoints, inputPoints, pointNumber*sizeof(Point), hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sp), inputPoints, 2000 * sizeof(Point)));

	hipEventRecord( memcopy, 0 );

//###############################################################

	dim3 threads; 
	threads.x = threadsNum;

    dim3 blocks; 
	blocks.x = ( ( allNum + threads.x - 1 ) / threads.x );

	printf ("thread = %d ; blocks %d \n", threads.x, blocks.x);

	//cudaCreateNeighborsTable<<<blocks, threads>>>(cudaPoints, pointNumber, eps, cudaNeighborhoodMatrix, minPoint);
	cudaCreateNeighborsTableShared<<<blocks, threads>>>(cudaPoints, pointNumber, eps, cudaNeighborhoodMatrix, minPoint);

	hipEventRecord( neighbor, 0 );

//##############################################################

	CUDA_CHECK_RETURN(hipMemcpy(hostNeighborhoodMatrix, cudaNeighborhoodMatrix, pointNumber*pointNumber*sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipMemcpy(inputPoints, cudaPoints, pointNumber*sizeof(Point), hipMemcpyDeviceToHost));

	hipEventRecord( mainCopy, 0 );

	/*for(int i = 0; i < pointNumber; i++)
	{
		for(int j =0;j<pointNumber;j++)
		{
			printf("%7.2f",hostDistTable[i*pointNumber + j]);
		}
		printf ("\n");
	}

	for(int i=0;i<pointNumber;i++)
	{
		printf("hint = %d\t", inputPoints[i].hint);
		for(int j =0;j<pointNumber;j++)
		{
			printf("%7.3f",hostNeighborhoodMatrix[i*pointNumber + j]);
		}
		printf ("\n");
	}*/

//###############################################

	clock_t hostRun = clock();
	hostLabeling(inputPoints, pointNumber, hostNeighborhoodMatrix);
    hostLabelingTime = clock() - hostRun;

//	cudaLabeling<<<blocks, threads>>>(cudaPoints, pointNumber, cudaNeighborhoodMatrixPower);
//
//	hipEventRecord( labeling, 0 );
//
//	CUDA_CHECK_RETURN(hipMemcpy(inputPoints, cudaPoints,
//			 pointNumber*sizeof(Point2D), hipMemcpyDeviceToHost));



//##############################################

	hipFree(cudaPoints);
	hipFree(cudaNeighborhoodMatrix);

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventRecord( sstop, 0 );
	hipEventSynchronize( sstop );

	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );
	printf( "Time of all (stop):%3.2f ms\n", elapsedTime );

	hipEventElapsedTime( &elapsedTime, start, sstop );
	printf( "Time of all (sstop):%3.2f ms\n", elapsedTime );

	hipEventElapsedTime( &elapsedTime, start, allocate );
	printf( "Time of allocate:%3.2f ms\n", elapsedTime );

	hipEventElapsedTime( &elapsedTime, allocate, memcopy );
	printf( "Time of memcopy:%3.2f ms\n", elapsedTime );

	hipEventElapsedTime( &elapsedTime, memcopy, neighbor );
	printf( "Time of neighbor matrix:%3.2f ms\n", elapsedTime );

	printf("Host labeling: %d clicks (%f seconds).\n", hostLabelingTime, ((float) hostLabelingTime)/CLOCKS_PER_SEC);
//	hipEventElapsedTime( &elapsedTime, labeling , mainCopy);
//	printf( "Time of mainCopy:%3.2f ms\n", elapsedTime );

	hipEventElapsedTime( &elapsedTime, mainCopy , stop);
	printf( "Time of free:%3.2f ms\n", elapsedTime );

	hipEventDestroy(start);
	hipEventDestroy(allocate);
	hipEventDestroy(memcopy);
	hipEventDestroy(labeling);
	hipEventDestroy(mainCopy);
	hipEventDestroy(stop);
	hipEventDestroy(sstop);

	return NULL;
}

